
#include <hip/hip_runtime.h>



#ifdef VERSION1
__global__ void ocean_kernel(int *grid, int xdim, int ydim, int offset)
{
    int threads = gridDim.x*blockDim.x;
    int threadId  = blockDim.x*blockIdx.x + threadIdx.x;

    if (threads > (xdim-2)*(ydim-2)) {
        threads = (xdim-2)*(ydim-2);
        if (threadId >= threads) {
            return;
        }
    }

    int chunk = (xdim-2)*(ydim-2)/threads;
    int start = threadId * chunk;
    int end = (threadId + 1) * chunk;

    int threadsPerRow = (xdim - 2);

    for (int i=start; i<end; i++) {
        if (offset) {
            if (i%2) continue;
        } else {
            if (!(i%2)) continue;
        }

        int row = i / threadsPerRow;
        int col = i % threadsPerRow;

        int loc = xdim + row * xdim + col;
        if (offset) {
            loc += (row%2) ? 1 : 0;
            loc += 1;
        } else {
            loc += (row%2) ? 0 : 1;
        }
        // printf("Row: %d, Col: %d\n", row, col);
        // printf("loc: %d\n", loc);

        grid[loc] = (grid[loc]
                  + grid[loc - xdim]
                  + grid[loc + xdim]
                  + grid[loc + 1]
                  + grid[loc - 1])
                  / 5;
    }
}
#endif


#ifdef VERSION2
__global__ void ocean_kernel(int *grid, int xdim, int ydim, int offset)
{
    int threads = gridDim.x*blockDim.x;
    int threadId  = blockDim.x*blockIdx.x + threadIdx.x;

    if (threads > (xdim-2)*(ydim-2)) {
        threads = (xdim-2)*(ydim-2);
        if (threadId >= threads) {
            return;
        }
    }

    int chunk = (xdim-2)*(ydim-2)/threads;
    int start = 0.;

    int threadsPerRow = (xdim - 2);

    for (int i=start; i < chunk; i++) {
      if (offset){
            if (threadIdx.x % 2) continue;
        } else {
            if (!(threadIdx.x % 2)) continue;
        }

        int row = (i * threads) / threadsPerRow;
        int col = (i * threads) % threadsPerRow;

        int loc = xdim + row * xdim + col + threadId;
        if (offset) {
            loc += (row%2) ? 1 : 0;
            loc += 1;
        } else {
            loc += (row%2) ? 0 : 1;
        }

        grid[loc] = (grid[loc]
                  + grid[loc - xdim]
                  + grid[loc + xdim]
                  + grid[loc + 1]
                  + grid[loc - 1])
                  / 5;
    }
}
#endif


#ifdef VERSION3
__global__ void ocean_kernel_V2(int *grid, int xdim, int ydim, int offset)
{
    int threads = gridDim.x*blockDim.x;
    int threadId  = blockDim.x*blockIdx.x + threadIdx.x;

    if (threads > (xdim-2)*(ydim-2)) {
        threads = (xdim-2)*(ydim-2);
        if (threadId >= threads) {
            return;
        }
    }

    int chunk = (xdim-2)*(ydim-2)/threads;
    int start = 0.;

    int threadsPerRow = (xdim - 2);

    for (int i=start; i < chunk; i++) {
      if (offset){
            if (threadIdx.x % 2) continue;
        } else {
            if (!(threadIdx.x % 2)) continue;
        }

        int row = (i * threads) / threadsPerRow;
        int col = (i * threads) % threadsPerRow;

        int loc = xdim + row * xdim + col + threadId;
        if (offset) {
            loc += (row%2) ? 1 : 0;
            loc += 1;
        } else {
            loc += (row%2) ? 0 : 1;
        }

        grid[loc] = (grid[loc]
                  + grid[loc - xdim]
                  + grid[loc + xdim]
                  + grid[loc + 1]
                  + grid[loc - 1])
                  / 5;
    }
}

__global__ void split_array_kernel(int *grid, int *red_grid, int *black_grid, int xdim, int ydim)
{
  // This kernel should take the contents of grid and copy all of the red
  // elements into red_grid and all of the black elements into black_grid

  int threads  = gridDim.x * blockDim.x;
  int threadId = blockDim.x * blockIdx.x + threadIdx.x;
  
  if (threads > (xdim - 2)*(ydim - 2) ) {
    threads = (xdim - 2) * (ydim - 2);
    if (threadId >= threads) {
      return;
    }
  }
  
  int chunk = (xdim - 2) * (ydim - 2) / threads;
  if( (xdim - 2) * (ydim - 2) % threads )
    return;

  int threadsPerRow = (xdim - 2);
  int red;

  for (int i = 0; i < chunk; i++) {
    int row = (i * threads + threadId) / threadsPerRow;
    int col = (i * threads + threadId) % threadsPerRow;
    int loc = (row + 1) * xdim + 1 + col;

    if(row % 2){
      if(col % 2)
	red = 0;
      else
	red = 1;
    }
    else{
      if(col % 2)
	red = 1;
      else
	red = 0;
    }

    if( red )
      red_grid[loc / 2] = grid[loc];
    else
      black_grid[loc / 2] = grid[loc];

    // BOUNDARIES
    // first row
    if(row == 0){
      if( red )
    	black_grid[col / 2 + 1] = grid[loc - xdim];
      else
    	red_grid[col / 2] = grid[loc - xdim];
    }
    // last row
    if(row == ydim - 3){
      if( red )
    	black_grid[(loc + xdim) / 2 ] = grid[loc + xdim];
      else
    	red_grid[(loc + xdim) / 2] = grid[loc + xdim];
    }

    // left column
    if(col == 0){
      if( red )
    	black_grid[loc / 2] = grid[loc - 1];
      else
    	red_grid[loc / 2] = grid[loc - 1];
    }

    // right column
    if(col == xdim - 3){
      if( red )
    	black_grid[loc / 2] = grid[loc + 1];
      else
    	red_grid[loc / 2] = grid[loc + 1];
    }
  }
}

__global__ void unsplit_array_kernel(int *grid, int *red_grid, int *black_grid, int xdim, int ydim)
{
  // "Inverse" of the previous function
  int threads  = gridDim.x * blockDim.x;
  int threadId = blockDim.x * blockIdx.x + threadIdx.x;
  
  if (threads > (xdim - 2)*(ydim - 2) ) {
    threads = (xdim - 2) * (ydim - 2);
    if (threadId >= threads) {
      return;
    }
  }
  
  int chunk = (xdim - 2) * (ydim - 2) / threads;
  if( (xdim - 2) * (ydim - 2) % threads )
    return;

  int threadsPerRow = (xdim - 2);
  int red;

  for (int i = 0; i < chunk; i++) {
    int row = (i * threads + threadId) / threadsPerRow;
    int col = (i * threads + threadId) % threadsPerRow;
    int loc = (row + 1) * xdim + 1 + col;

    if(row % 2){
      if(col % 2)
	red = 0;
      else
	red = 1;
    }
    else{
      if(col % 2)
	red = 1;
      else
	red = 0;
    }

    if( red )
      grid[loc] = red_grid[loc / 2];
    else
      grid[loc] = black_grid[loc / 2];

    // BOUNDARIES
    if(row == 0){
      if( red )
    	grid[loc - xdim] = black_grid[col / 2 + 1];
      else
    	grid[loc - xdim] = red_grid[col / 2];
    }
    // last row
    if(row == ydim - 3){
      if( red )
    	grid[loc + xdim] = black_grid[(loc + xdim) / 2 ];
      else
    	grid[loc + xdim] = red_grid[(loc + xdim) / 2];
    }

    // left column
    if(col == 0){
      if( red )
    	grid[loc - 1] = black_grid[loc / 2];
      else
    	grid[loc - 1] = red_grid[loc / 2];
    }

    // right column
    if(col == xdim - 3){
      if( red )
    	grid[loc + 1] = black_grid[loc / 2];
      else
    	grid[loc + 1] = red_grid[loc / 2];
    }
  }
}

__global__ void ocean_kernel(int *red_grid, int *black_grid, int xdim, int ydim, int offset)
{
  int threads  = gridDim.x * blockDim.x;
  int threadId = blockDim.x * blockIdx.x + threadIdx.x;

  if (threads > ((xdim - 2) * (ydim - 2) / 2)) {
    threads = (xdim - 2) * (ydim - 2) / 2;
    if (threadId >= threads)
      return;
  }

  if( (xdim - 2) * (ydim - 2) % (2 * threads) )
    return;

  int chunk = (xdim - 2) * (ydim - 2) / (2 * threads);
  int threadsPerRow = (xdim - 2) / 2;
  int i, edge, left, right;
  int shift = xdim / 2;

  for(i = 0; i < chunk; i++){

    int row = (i * threads + threadId) / threadsPerRow;
    int loc = threadId + i * threads + shift;

    if(offset){
      //black_grid update
      edge = (row + 1) / 2;
      edge *= 2;
      loc  += edge; // need to skip the elements in the boundaries
      if( row % 2 ){
	left = loc - 1;
	right = loc;
      }
      else{
	left = loc;
	right = loc + 1;
      }

      black_grid[loc] = (black_grid[loc]
			 + red_grid[loc - shift]
			 + red_grid[loc + shift]
			 + red_grid[left]
			 + red_grid[right]) / 5;
    }
    else{
      // red_grid update
      loc += 1;
      edge = row / 2;
      edge *= 2;
      loc  += edge; // need to skip the elements in the boundaries

      if( row % 2 ){
	left = loc;
	right = loc + 1;
      }
      else{
	left = loc - 1;
	right = loc;
      }

      red_grid[loc] = (red_grid[loc]
		       + black_grid[loc - shift]
		       + black_grid[loc + shift]
		       + black_grid[left]
		       + black_grid[right]) / 5;

    }

  }
}

#endif // VERSION3

#ifdef VERSION33

__global__ void split_array_kernel(int *grid, int *red_grid, int *black_grid, int xdim, int ydim)
{
  // This kernel should take the contents of grid and copy all of the red
  // elements into red_grid and all of the black elements into black_grid

  int threads  = gridDim.x*blockDim.x;
  int threadId = blockDim.x*blockIdx.x + threadIdx.x;
  
  if (threads > (xdim - 2)*(ydim - 2) ) {
    threads = (xdim - 2) * (ydim - 2);
    if (threadId >= threads) {
      return;
    }
  }
  
  int chunk = (xdim-2)*(ydim-2)/threads;
  int start = 0.;
  int end = chunk;
  int last_row = 0;

  int threadsPerRow = (xdim - 2);
  int par = threadIdx.x;

  for (int i=start; i < end; i++) {
    int row = (i * threads) / threadsPerRow;
    int col = (i * threads) % threadsPerRow;
    int loc = xdim + row * xdim + col + threadId + 1;

    if( par % 2 )
      red_grid[loc / 2 + xdim / 2 + 1] = grid[loc];

    else
      black_grid[loc / 2 + xdim / 2] = grid[loc];

    // BOUNDARIES
    // first row
    if(loc < threadsPerRow){
      if( par % 2 )
    	black_grid[threadIdx.x + i * threadsPerRow / threads] = grid[loc - xdim];
      else
    	red_grid[threadIdx.x + i * threadsPerRow / threads] = grid[loc - xdim];
    }
    // last row
    if(loc > xdim * (ydim - 2)){
      if( par % 2 )
    	black_grid[xdim*ydim/2 - xdim / 2 + threadIdx.x + last_row * threadsPerRow/ threads] = grid[loc + xdim];
      else
    	red_grid[xdim*ydim/2 - xdim / 2 + threadIdx.x + last_row * threadsPerRow / threads] = grid[loc + xdim];

      last_row++;
    }

    // left column
    if(loc % (xdim + 1) == 0){
      if(par % 2)
    	black_grid[xdim * (1 + row) / 2] = grid[loc - 1];
      else
    	red_grid[xdim * (1 + row) / 2] = grid[loc - 1];
    }

    // right column
    if(loc % (xdim - 1) == 0){
      if(par % 2)
    	black_grid[xdim * (2 + row) / 2 - 1] = grid[loc - 1];
      else
    	red_grid[xdim * (2 + row) / 2 - 1] = grid[loc - 1];
    }

    // THIS WORK ONLY IF threads is multiple of threadsPerRow && threads % 2 == 0
    // the idea is that in the next row each thread will upload a
    // different array (red_grid / black_grid) with respect to the previous one
    if( i % threadsPerRow / threads == 0 && i > 0)
      par++;
  }
}

__global__ void unsplit_array_kernel(int *grid, int *red_grid, int *black_grid, int xdim, int ydim)
{
    // This kernel should take the red_grid and black_grid and copy it back into grid
}

__global__ void ocean_kernel(int *red_grid, int *black_grid, int xdim, int ydim, int offset)
{
    // Your code for step 3
}
#endif

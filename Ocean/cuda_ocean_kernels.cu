
#include <hip/hip_runtime.h>



#ifdef VERSION1
__global__ void ocean_kernel(int *grid, int xdim, int ydim, int offset)
{
    int threads = gridDim.x*blockDim.x;
    int threadId  = blockDim.x*blockIdx.x + threadIdx.x;

    if (threads > (xdim-2)*(ydim-2)) {
        threads = (xdim-2)*(ydim-2);
        if (threadId >= threads) {
            return;
        }
    }

    int chunk = (xdim-2)*(ydim-2)/threads;
    int start = threadId * chunk;
    int end = (threadId + 1) * chunk;

    int threadsPerRow = (xdim - 2);

    for (int i=start; i<end; i++) {
        if (offset) {
            if (i%2) continue;
        } else {
            if (!(i%2)) continue;
        }

        int row = i / threadsPerRow;
        int col = i % threadsPerRow;

        int loc = xdim + row * xdim + col;
        if (offset) {
            loc += (row%2) ? 1 : 0;
            loc += 1;
        } else {
            loc += (row%2) ? 0 : 1;
        }
        // printf("Row: %d, Col: %d\n", row, col);
        // printf("loc: %d\n", loc);

        grid[loc] = (grid[loc]
                  + grid[loc - xdim]
                  + grid[loc + xdim]
                  + grid[loc + 1]
                  + grid[loc - 1])
                  / 5;
    }
}
#endif


#ifdef VERSION2
__global__ void ocean_kernel(int *grid, int xdim, int ydim, int offset)
{
    int threads = gridDim.x*blockDim.x;
    int threadId  = blockDim.x*blockIdx.x + threadIdx.x;

    if (threads > (xdim-2)*(ydim-2)) {
        threads = (xdim-2)*(ydim-2);
        if (threadId >= threads) {
            return;
        }
    }

    int chunk = (xdim-2)*(ydim-2)/threads;
    int start = 0.;

    int threadsPerRow = (xdim - 2);

    for (int i=start; i < chunk; i++) {
      if (offset){
            if (threadIdx.x % 2) continue;
        } else {
            if (!(threadIdx.x % 2)) continue;
        }

        int row = (i * threads) / threadsPerRow;
        int col = (i * threads) % threadsPerRow;

        int loc = xdim + row * xdim + col + threadId;
        if (offset) {
            loc += (row%2) ? 1 : 0;
            loc += 1;
        } else {
            loc += (row%2) ? 0 : 1;
        }

        grid[loc] = (grid[loc]
                  + grid[loc - xdim]
                  + grid[loc + xdim]
                  + grid[loc + 1]
                  + grid[loc - 1])
                  / 5;
    }
}
#endif


#ifdef VERSION3

__global__ void split_array_kernel(int *grid, int *red_grid, int *black_grid, int xdim, int ydim)
{
  // This kernel should take the contents of grid and copy all of the red
  // elements into red_grid and all of the black elements into black_grid

  int threads  = gridDim.x * blockDim.x;
  int threadId = blockDim.x * blockIdx.x + threadIdx.x;
  
  if (threads > (xdim - 2)*(ydim - 2) ) {
    threads = (xdim - 2) * (ydim - 2);
    if (threadId >= threads) {
      return;
    }
  }
  
  int chunk = (xdim - 2) * (ydim - 2) / threads;
  if( (xdim - 2) * (ydim - 2) % threads )
    return;

  int threadsPerRow = (xdim - 2);
  int red;

  for (int i = 0; i < chunk; i++) {
    int row = (i * threads + threadId) / threadsPerRow;
    int col = (i * threads + threadId) % threadsPerRow;
    int loc = (row + 1) * xdim + 1 + col;

    if(row % 2){
      if(col % 2)
	red = 0;
      else
	red = 1;
    }
    else{
      if(col % 2)
	red = 1;
      else
	red = 0;
    }

    if( red )
      red_grid[loc / 2] = grid[loc];
    else
      black_grid[loc / 2] = grid[loc];

    // BOUNDARIES
    if(row == 0){
      if( red )
    	black_grid[col / 2 + 1] = grid[loc - xdim];
      else
    	red_grid[col / 2] = grid[loc - xdim];
    }
    // last row
    if(row == ydim - 3){
      if( red )
    	black_grid[(loc + xdim) / 2 ] = grid[loc + xdim];
      else
    	red_grid[(loc + xdim) / 2] = grid[loc + xdim];
    }

    // left column
    if(col == 0){
      if( red )
    	black_grid[loc / 2] = grid[loc - 1];
      else
    	red_grid[loc / 2] = grid[loc - 1];
    }

    // right column
    if(col == xdim - 3){
      if( red )
    	black_grid[loc / 2] = grid[loc + 1];
      else
    	red_grid[loc / 2] = grid[loc + 1];
    }
  }
}

__global__ void unsplit_array_kernel(int *grid, int *red_grid, int *black_grid, int xdim, int ydim)
{
    // This kernel should take the red_grid and black_grid and copy it back into grid
}

__global__ void ocean_kernel(int *red_grid, int *black_grid, int xdim, int ydim, int offset)
{
    // Your code for step 3
}

#endif // VERSION3

#ifdef VERSION33

__global__ void split_array_kernel(int *grid, int *red_grid, int *black_grid, int xdim, int ydim)
{
  // This kernel should take the contents of grid and copy all of the red
  // elements into red_grid and all of the black elements into black_grid

  int threads  = gridDim.x*blockDim.x;
  int threadId = blockDim.x*blockIdx.x + threadIdx.x;
  
  if (threads > (xdim - 2)*(ydim - 2) ) {
    threads = (xdim - 2) * (ydim - 2);
    if (threadId >= threads) {
      return;
    }
  }
  
  int chunk = (xdim-2)*(ydim-2)/threads;
  int start = 0.;
  int end = chunk;
  int last_row = 0;

  int threadsPerRow = (xdim - 2);
  int par = threadIdx.x;

  for (int i=start; i < end; i++) {
    int row = (i * threads) / threadsPerRow;
    int col = (i * threads) % threadsPerRow;
    int loc = xdim + row * xdim + col + threadId + 1;

    if( par % 2 )
      red_grid[loc / 2 + xdim / 2 + 1] = grid[loc];

    else
      black_grid[loc / 2 + xdim / 2] = grid[loc];

    // BOUNDARIES
    // first row
    if(loc < threadsPerRow){
      if( par % 2 )
    	black_grid[threadIdx.x + i * threadsPerRow / threads] = grid[loc - xdim];
      else
    	red_grid[threadIdx.x + i * threadsPerRow / threads] = grid[loc - xdim];
    }
    // last row
    if(loc > xdim * (ydim - 2)){
      if( par % 2 )
    	black_grid[xdim*ydim/2 - xdim / 2 + threadIdx.x + last_row * threadsPerRow/ threads] = grid[loc + xdim];
      else
    	red_grid[xdim*ydim/2 - xdim / 2 + threadIdx.x + last_row * threadsPerRow / threads] = grid[loc + xdim];

      last_row++;
    }

    // left column
    if(loc % (xdim + 1) == 0){
      if(par % 2)
    	black_grid[xdim * (1 + row) / 2] = grid[loc - 1];
      else
    	red_grid[xdim * (1 + row) / 2] = grid[loc - 1];
    }

    // right column
    if(loc % (xdim - 1) == 0){
      if(par % 2)
    	black_grid[xdim * (2 + row) / 2 - 1] = grid[loc - 1];
      else
    	red_grid[xdim * (2 + row) / 2 - 1] = grid[loc - 1];
    }

    // THIS WORK ONLY IF threads is multiple of threadsPerRow && threads % 2 == 0
    // the idea is that in the next row each thread will upload a
    // different array (red_grid / black_grid) with respect to the previous one
    if( i % threadsPerRow / threads == 0 && i > 0)
      par++;
  }
}

__global__ void unsplit_array_kernel(int *grid, int *red_grid, int *black_grid, int xdim, int ydim)
{
    // This kernel should take the red_grid and black_grid and copy it back into grid
}

__global__ void ocean_kernel(int *red_grid, int *black_grid, int xdim, int ydim, int offset)
{
    // Your code for step 3
}
#endif

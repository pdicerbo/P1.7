
#include <hip/hip_runtime.h>



#ifdef VERSION1
__global__ void ocean_kernel(int *grid, int xdim, int ydim, int offset)
{
    int threads = gridDim.x*blockDim.x;
    int threadId  = blockDim.x*blockIdx.x + threadIdx.x;

    if (threads > (xdim-2)*(ydim-2)) {
        threads = (xdim-2)*(ydim-2);
        if (threadId >= threads) {
            return;
        }
    }

    int chunk = (xdim-2)*(ydim-2)/threads;
    int start = threadId * chunk;
    int end = (threadId + 1) * chunk;

    int threadsPerRow = (xdim - 2);

    for (int i=start; i<end; i++) {
        if (offset) {
            if (i%2) continue;
        } else {
            if (!(i%2)) continue;
        }

        int row = i / threadsPerRow;
        int col = i % threadsPerRow;

        int loc = xdim + row * xdim + col;
        if (offset) {
            loc += (row%2) ? 1 : 0;
            loc += 1;
        } else {
            loc += (row%2) ? 0 : 1;
        }
        // printf("Row: %d, Col: %d\n", row, col);
        // printf("loc: %d\n", loc);

        grid[loc] = (grid[loc]
                  + grid[loc - xdim]
                  + grid[loc + xdim]
                  + grid[loc + 1]
                  + grid[loc - 1])
                  / 5;
    }
}
#endif


#ifdef VERSION2
__global__ void ocean_kernel(int *grid, int xdim, int ydim, int offset)
{
    int threads = gridDim.x*blockDim.x;
    int threadId  = blockDim.x*blockIdx.x + threadIdx.x;

    if (threads > (xdim-2)*(ydim-2)) {
        threads = (xdim-2)*(ydim-2);
        if (threadId >= threads) {
            return;
        }
    }

    int chunk = (xdim-2)*(ydim-2)/threads;
    int start = 0.;

    int threadsPerRow = (xdim - 2);

    for (int i=start; i < chunk; i++) {
      if (offset){
            if (threadIdx.x % 2) continue;
        } else {
            if (!(threadIdx.x % 2)) continue;
        }

        int row = (i * threads) / threadsPerRow;
        int col = (i * threads) % threadsPerRow;

        int loc = xdim + row * xdim + col + threadId;
        if (offset) {
            loc += (row%2) ? 1 : 0;
            loc += 1;
        } else {
            loc += (row%2) ? 0 : 1;
        }

        grid[loc] = (grid[loc]
                  + grid[loc - xdim]
                  + grid[loc + xdim]
                  + grid[loc + 1]
                  + grid[loc - 1])
                  / 5;
    }
}
#endif


#ifdef VERSION3

__global__ void split_array_kernel(int *grid, int *red_grid, int *black_grid, int xdim, int ydim)
{
  // This kernel should take the contents of grid and copy all of the red
  // elements into red_grid and all of the black elements into black_grid

  int threads  = gridDim.x * blockDim.x;
  int threadId = blockDim.x * blockIdx.x + threadIdx.x;
  
  if (threads > (xdim - 2)*(ydim - 2) ) {
    threads = (xdim - 2) * (ydim - 2);
    if (threadId >= threads) {
      return;
    }
  }
  
  int chunk = (xdim - 2) * (ydim - 2) / threads;
  if( (xdim - 2) * (ydim - 2) % threads )
    return;

  int threadsPerRow = (xdim - 2);
  int red;

  for (int i = 0; i < chunk; i++) {
    int row = (i * threads + threadId) / threadsPerRow;
    int col = (i * threads + threadId) % threadsPerRow;
    int loc = (row + 1) * xdim + 1 + col;

    if(row % 2){
      if(col % 2)
	red = 0;
      else
	red = 1;
    }
    else{
      if(col % 2)
	red = 1;
      else
	red = 0;
    }

    if( red )
      red_grid[loc / 2] = grid[loc];
    else
      black_grid[loc / 2] = grid[loc];

    // BOUNDARIES
    // first row
    if(row == 0){
      if( red )
    	black_grid[col / 2 + 1] = grid[loc - xdim];
      else
    	red_grid[col / 2] = grid[loc - xdim];
    }
    // last row
    if(row == ydim - 3){
      if( red )
    	black_grid[(loc + xdim) / 2 ] = grid[loc + xdim];
      else
    	red_grid[(loc + xdim) / 2] = grid[loc + xdim];
    }

    // left column
    if(col == 0){
      if( red )
    	black_grid[loc / 2] = grid[loc - 1];
      else
    	red_grid[loc / 2] = grid[loc - 1];
    }

    // right column
    if(col == xdim - 3){
      if( red )
    	black_grid[loc / 2] = grid[loc + 1];
      else
    	red_grid[loc / 2] = grid[loc + 1];
    }
  }
}

__global__ void unsplit_array_kernel(int *grid, int *red_grid, int *black_grid, int xdim, int ydim)
{
  // "Inverse" of the previous function
  int threads  = gridDim.x * blockDim.x;
  int threadId = blockDim.x * blockIdx.x + threadIdx.x;
  
  if (threads > (xdim - 2)*(ydim - 2) ) {
    threads = (xdim - 2) * (ydim - 2);
    if (threadId >= threads) {
      return;
    }
  }
  
  int chunk = (xdim - 2) * (ydim - 2) / threads;
  if( (xdim - 2) * (ydim - 2) % threads )
    return;

  int threadsPerRow = (xdim - 2);
  int red;

  for (int i = 0; i < chunk; i++) {
    int row = (i * threads + threadId) / threadsPerRow;
    int col = (i * threads + threadId) % threadsPerRow;
    int loc = (row + 1) * xdim + 1 + col;

    if(row % 2){
      if(col % 2)
	red = 0;
      else
	red = 1;
    }
    else{
      if(col % 2)
	red = 1;
      else
	red = 0;
    }

    if( red )
      grid[loc] = red_grid[loc / 2];
    else
      grid[loc] = black_grid[loc / 2];

    // BOUNDARIES
    if(row == 0){
      if( red )
    	grid[loc - xdim] = black_grid[col / 2 + 1];
      else
    	grid[loc - xdim] = red_grid[col / 2];
    }
    // last row
    if(row == ydim - 3){
      if( red )
    	grid[loc + xdim] = black_grid[(loc + xdim) / 2 ];
      else
    	grid[loc + xdim] = red_grid[(loc + xdim) / 2];
    }

    // left column
    if(col == 0){
      if( red )
    	grid[loc - 1] = black_grid[loc / 2];
      else
    	grid[loc - 1] = red_grid[loc / 2];
    }

    // right column
    if(col == xdim - 3){
      if( red )
    	grid[loc + 1] = black_grid[loc / 2];
      else
    	grid[loc + 1] = red_grid[loc / 2];
    }
  }
}

__global__ void ocean_kernel(int *red_grid, int *black_grid, int xdim, int ydim, int offset)
{
  int threads  = gridDim.x * blockDim.x;
  int threadId = blockDim.x * blockIdx.x + threadIdx.x;

  if (threads > ((xdim - 2) * (ydim - 2) / 2)) {
    threads = (xdim - 2) * (ydim - 2) / 2;
    if (threadId >= threads)
      return;
  }

  if( (xdim - 2) * (ydim - 2) % (2 * threads) )
    return;

  int chunk = (xdim - 2) * (ydim - 2) / (2 * threads);
  int threadsPerRow = (xdim - 2) / 2;
  int i, edge, left, right;
  int shift = xdim / 2;

  for(i = 0; i < chunk; i++){

    int row = (i * threads + threadId) / threadsPerRow;
    int loc = threadId + i * threads + shift;

    if(offset){
      //black_grid update
      edge = (row + 1) / 2;
      edge *= 2;
      loc  += edge; // need to skip the elements in the boundaries
      if( row & 1 ){
	left = loc - 1;
	right = loc;
      }
      else{
	left = loc;
	right = loc + 1;
      }

      black_grid[loc] = (black_grid[loc]
			 + red_grid[loc - shift]
			 + red_grid[loc + shift]
			 + red_grid[left]
			 + red_grid[right]) / 5;
    }
    else{
      // red_grid update
      loc += 1;
      edge = row / 2;
      edge *= 2;
      loc  += edge; // need to skip the elements in the boundaries

      if( row & 1 ){
	left = loc;
	right = loc + 1;
      }
      else{
	left = loc - 1;
	right = loc;
      }

      red_grid[loc] = (red_grid[loc]
		       + black_grid[loc - shift]
		       + black_grid[loc + shift]
		       + black_grid[left]
		       + black_grid[right]) / 5;
    }

  }
}

#endif // VERSION3